#include "cuda_measurable_event.cuh"
#include "macros.cuh"

CudaMeasurableEvent::CudaMeasurableEvent(const std::string& name)
    : mName(name)
{
    CUDA_CHECK_PRINT(hipEventCreate(&mStart));
    CUDA_CHECK_PRINT(hipEventCreate(&mStop));
    CUDA_CHECK_PRINT(hipEventRecord(mStart));
}

CudaMeasurableEvent::~CudaMeasurableEvent()
{
    CUDA_CHECK_PRINT(hipEventRecord(mStop));
    CUDA_CHECK_PRINT(hipEventSynchronize(mStop));
    float milliseconds = 0;
    CUDA_CHECK_PRINT(hipEventElapsedTime(&milliseconds, mStart, mStop));

    auto& stats = Statistics::get();
    stats.addSample(mName, milliseconds);

    CUDA_CHECK_PRINT(hipEventDestroy(mStart));
    CUDA_CHECK_PRINT(hipEventDestroy(mStop));
}
